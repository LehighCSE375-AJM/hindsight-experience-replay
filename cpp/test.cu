#include "hip/hip_runtime.h"
#include <iostream>
#include "tensor.h"
#include "cuda_utils.h"

using namespace std;

__global__ void matrixAddOne(Tensor m, Tensor out) {
	// m.tanh(out); and m.copy(out); also work
	m.relu(out);
	// Something that we have to think about is doing a __syncthreads() before we do not 
	// elementwise operations such as matrix multiplication (not needed for element-wise 
	// operations since they always act on the same value as the last element-wise operation
	// so they will be consistent). There is a chance that one warp is still finishing the 
	// previous relu, while another thread is doing matrix multiplication with that value. 
	// (something interesting is that newer gpu's allow different threads in the same warp to
	// run operations simultaneously so in theory this could be an issue within a single war
	// too)

	// Another thing to consider if multiple element operations are done in a row then it 
	// could make sense to do it like this: bring subset of matrix into shared memory, 
	// perform operations, write back to global memory, then operate on next subset of 
	// matrix. Would be instead of: read all of matrix from global memory, operate on, then 
	// write back to global memory then do this for the next operation. Ends up as 1 
	// read/write from global memory for all of the operations instead of 1 read/write per 
	// operation from global memory. This may be significantly faster since reading from 
	// shared memory is ~10x faster than global even with coalescense. 

	// Very helpful that we can print from a kernel
	out.print();
}

int main() {
	Tensor t(10, 10, [](int i) {return i - 20;});
	cout << t << endl;
	t.cudafy();
	Tensor out(10, 5);
	out.cudafy();
	matrixAddOne<<<1, 4>>>(t, out);
	gpuErrchk(hipDeviceSynchronize());
	// This uncudafy doesn't work since the cuda memory address was updated by the kernel
	// (pass-by-value so not updated on the host. Can't be pass-by-reference since then the 
	// kernel would be reading host memory) It doesn't really matter that it doesn't work btw
	// since most (all?) of the tensors will stay within the kernel. (for that same reason 
	// the cudafy method probably isn't needed either)
	out.uncudafy();
}
