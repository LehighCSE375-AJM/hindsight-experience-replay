#include "hip/hip_runtime.h"
#include <iostream>
#include "tensor.h"
#include "cuda_utils.h"
#include <hiprand/hiprand_kernel.h>
#include "linear.h"

using namespace std;

__global__ void matrixAddOne(unsigned long seed) {
	hiprandState rand_state;
        hiprand_init(seed, threadIdx.x, 0, &rand_state);
	Linear lin(4, 2, RELU, rand_state);
	Tensor in = Tensor(1, 4);
	in.values[0] = 8;
	in.values[1] = 2;
	in.values[2] = 9;
	in.print("In Matrix times");
	Tensor out = lin.forward(in);
	out.print("Out/Error Matrix");
	
	Tensor error;
	out.transpose(error);
	// We'll just use the output as the error for testing
	Tensor grad = lin.compute_gradient(error);
	grad.print("Gradient");

	// Something that we have to think about is doing a __syncthreads() before we do not 
	// elementwise operations such as matrix multiplication (not needed for element-wise 
	// operations since they always act on the same value as the last element-wise operation
	// so they will be consistent). There is a chance that one warp is still finishing the 
	// previous relu, while another thread is doing matrix multiplication with that value. 
	// (something interesting is that newer gpu's allow different threads in the same warp to
	// run operations simultaneously so in theory this could be an issue within a single war
	// too)

	// Another thing to consider if multiple element operations are done in a row then it 
	// could make sense to do it like this: bring subset of matrix into shared memory, 
	// perform operations, write back to global memory, then operate on next subset of 
	// matrix. Would be instead of: read all of matrix from global memory, operate on, then 
	// write back to global memory then do this for the next operation. Ends up as 1 
	// read/write from global memory for all of the operations instead of 1 read/write per 
	// operation from global memory. This may be significantly faster since reading from 
	// shared memory is ~10x faster than global even with coalescense. 
	// This above idea might not be super helpful when we have to store the intermediate matrixes
	// but I would guess it will be good for the adam optimizer. (not really sure though, could 
	// just make sense to do everything in one call, idk)

	// Very helpful that we can print from a kernel
}

int main() {
	matrixAddOne<<<1, THREADS>>>(1234);
	gpuErrchk(hipDeviceSynchronize());
	// This uncudafy doesn't work since the cuda memory address was updated by the kernel
	// (pass-by-value so not updated on the host. Can't be pass-by-reference since then the 
	// kernel would be reading host memory) It doesn't really matter that it doesn't work btw
	// since most (all?) of the tensors will stay within the kernel. (for that same reason 
	// the cudafy method probably isn't needed either)
}
